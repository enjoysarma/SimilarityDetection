#include "hip/hip_runtime.h"
/**
 *
 * findMatchesKernel.cu - GPU kernels and wrapper functions are defined in this file
 *
 */
#include "Refactor.h"

#include <cmath>
#include <cstdint>
#include <iostream>

#define MAX_THREADS_PER_BLOCK 1024

/**
 * @brief: Error checking utility
 */
#define gpuErrorCheck(ans) {gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        std::cerr << "GPUassert: " << hipGetErrorString(code) << ' ' << file << ' ' << line << std::endl;
        if (abort) exit(code);
    }
}

/**
 * @brief: Convenience structure for input arguments passed to the GPU kernels
 */
template <typename T>
struct KernelVector
{
    T*  m_vector;
    T*  m_hashes;
    uint32_t m_size;

    KernelVector(T* dVec)
    {
        m_vector = dVec;
        m_size = 0;
    }

    KernelVector(T* dVec, T dVecSize)
    {
        m_vector 	= dVec;
        m_size 		= dVecSize;
    }

    KernelVector(T* dVec, T* dVecHashes, T dVecSize)
    {
        m_vector 	= dVec;
        m_hashes 	= dVecHashes;
        m_size 		= dVecSize;
    }
};


__device__ uint32_t compressTile(uint32_t fpi, uint32_t spi, uint32_t len)
{
    return (((fpi << TILE_FPI_SH) & MASK_TILE_FPI) |
            ((spi << TILE_SPI_SH) & MASK_TILE_SPI) |
            (len & MASK_TILE_LEN)
           );
}

//using __ldg() to cache read-only data from GMEM - this speeds up the kernel execution by near 12%
__global__ void findMatchesKernel_firstIter (
        KernelVector<uint32_t> dpaTokens,
        KernelVector<uint32_t> dpbTokens,
        uint32_t MML,
        KernelVector<uint32_t> dMatches,
        uint32_t* dMatchesIndex
        )
{
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    uint32_t substrLen = 0;

    // if (dpaTokens.m_hashes[idx] == dpbTokens.m_hashes[idy])
    if (__ldg(&dpaTokens.m_hashes[idx]) == __ldg(&dpbTokens.m_hashes[idy]))
    {
        while ((idx + substrLen < dpaTokens.m_size) &&
                (idy + substrLen < dpbTokens.m_size) &&
                (__ldg(&dpaTokens.m_vector[idx + substrLen]) == __ldg(&dpbTokens.m_vector[idy + substrLen]))
                // (dpaTokens.m_vector[idx + substrLen] == dpbTokens.m_vector[idy + substrLen])
              )
            substrLen++;		

        if (substrLen >= MML)
        {
            dMatches.m_vector[atomicAdd(dMatchesIndex, 1)] = compressTile(idx, idy, substrLen);
        }
    }

    return;
}

//using __ldg() to cache read-only data from GMEM - this speeds up the kernel execution by near 12%
__global__ void findMatchesKernel (
        KernelVector<uint32_t> dpaTokens,
        KernelVector<uint32_t> dpbTokens,
        uint32_t MML,
        KernelVector<uint32_t> dMatches,
        uint32_t* dMatchesIndex
        )
{
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    uint32_t substrLen = 0;

    
    // if (dpaTokens.m_hashes[idx] == dpbTokens.m_hashes[idy])
    if (__ldg(&dpaTokens.m_hashes[idx]) == __ldg(&dpbTokens.m_hashes[idy]))
    {
        while ((idx + substrLen < dpaTokens.m_size) &&
                (idy + substrLen < dpbTokens.m_size) &&
                (__ldg(&dpaTokens.m_vector[idx + substrLen]) == __ldg(&dpbTokens.m_vector[idy + substrLen]))
                // (dpaTokens.m_vector[idx + substrLen] == dpbTokens.m_vector[idy + substrLen])
              )
            substrLen++;		

        if (substrLen >= MML)
        {
            //following doesn't work in the case where no threads write to the output - input files are completely different
            // uint32_t* prevIndex = (dMatchesIndex - 1);
            // atomicCAS(dMatchesIndex, 0, *(prevIndex));
            
            dMatches.m_vector[atomicAdd(dMatchesIndex, 1)] = compressTile(idx, idy, substrLen);
        }
    }

    return;
}

/**
 *
 * @brief: Wrapper function around GPU kernels
 *
 * @param dpaTokens         - vector of tokens for the first program in comparison
 * @param dpaTokensHashes   - vector of hashes for the first program in comparison
 * @param dpaTokensSize     - sizeof(first_vector), needed for kernel grid and block size calculation
 * @param dpbTokens         - same as for the first program
 * @param dpbTokensHashes   - same as for the first program
 * @param dpbTokensSize     - same as for the first program
 * @param MML               - Minimum Match Length as predetermined in Parameters.cpp
 * @param dMatches          - output vector for storing results
 * @param dMatchesIndex     - index thats GPU threads use when writing to the output vector
 * @param iteration         - based on the value, different kernel is executed (first iteration is different from others)
 */
void findMatches(
    uint32_t*   dpaTokens,
    uint32_t*   dpaTokensHashes,
    uint32_t    dpaTokensSize,
    uint32_t*   dpbTokens,
    uint32_t*   dpbTokensHashes,
    uint32_t    dpbTokensSize,
    uint32_t    MML,
    uint32_t*   dMatches,
    uint32_t*   dMatchesIndex,
    uint32_t    iteration
)
{

    //threads are organized in a 2D grid,
    //this way each thread works on one pair of partA_hash and partB_hash
    dim3 dimBlock(sqrt(MAX_THREADS_PER_BLOCK), sqrt(MAX_THREADS_PER_BLOCK));
    dim3 dimGrid(
            ceil((dpaTokensSize - MML + 0.0) / dimBlock.x),
            ceil((dpbTokensSize - MML + 0.0) / dimBlock.y)
            );

    //currently both kernels execute same code
    if (0 == iteration)
    {
        findMatchesKernel_firstIter <<< dimGrid, dimBlock >>>
            (
             KernelVector<uint32_t>(dpaTokens, dpaTokensHashes, dpaTokensSize),
             KernelVector<uint32_t>(dpbTokens, dpbTokensHashes, dpbTokensSize),
             MML,
             dMatches,
             dMatchesIndex
            );        
    }
    else
    {
        findMatchesKernel <<< dimGrid, dimBlock >>>
            (
             KernelVector<uint32_t>(dpaTokens, dpaTokensHashes, dpaTokensSize),
             KernelVector<uint32_t>(dpbTokens, dpbTokensHashes, dpbTokensSize),
             MML,
             dMatches,
             dMatchesIndex
            );
    }
        
    gpuErrorCheck(hipPeekAtLastError());
}
